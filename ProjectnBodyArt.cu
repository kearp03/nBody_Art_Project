#include "hip/hip_runtime.h"
// nvcc nBodyArtB.cu -o nBodyArt -lglut -lm -lGLU -lGL
//To stop hit "control c" in the window you launched it from.
#include <iostream>
#include <fstream>
#include <sstream>
#include <string.h>
#include <GL/glut.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
using namespace std;

#define PI 3.141592653589793238462643383279502884197169399375105820974944592307816406286

FILE* ffmpeg;

// defines for terminal stuff.
#define BOLD_ON  "\e[1m"
#define BOLD_OFF   "\e[m"

FILE* MovieFile;

// Globals
int NumberOfBodies;
float TotalRunTime;
float Dt;
float G;
float H;
float Epsilon;
float MassOfBody;
float DiameterOfBody;
float VelocityMax;
float Drag;
int DrawRate;
int PrintRate;

// Other Globals
int Pause;
float *BodyPositionX, *BodyPositionY, *BodyPositionZ;
float *BodyVelocityX, *BodyVelocityY, *BodyVelocityZ;
float *BodyForceX, *BodyForceY, *BodyForceZ;
float *BodyColorX, *BodyColorY, *BodyColorZ;
int DrawTimer, PrintTimer;
float RunTime;
int* Buffer;
int MovieOn;
int MovieFlag;
int Trace;
double MouseX, MouseY, MouseZ;

// Window globals
static int Window;
int XWindowSize;
int YWindowSize; 
double Near;
double Far;
double EyeX;
double EyeY;
double EyeZ;
double CenterX;
double CenterY;
double CenterZ;
double UpX;
double UpY;
double UpZ;

// Prototyping functions
void setSimulationParameters();
void allocateMemory();
void setInitailConditions();
void drawPicture();
void nBody();
void errorCheck(const char*);
void terminalPrint();
void setup();
void movieOn();
void movieOff();
void screenShot();
float4 centerOfMass();
float4 linearVelocity();
void zeroOutSystem();

//#include "./callBackFunctions.h"

void Display()
{
	glClear(GL_COLOR_BUFFER_BIT);
	glClear(GL_DEPTH_BUFFER_BIT);
	drawPicture();
}

void idle()
{
	nBody();
}

void reshape(int w, int h)
{
	glViewport(0, 0, (GLsizei) w, (GLsizei) h);
}

void KeyPressed(unsigned char key, int x, int y)
{	
	if(key == 'q')
	{
		pclose(ffmpeg);
		glutDestroyWindow(Window);
		printf("\nw Good Bye\n");
		exit(0);
	}
	if(key == 'o')
	{
		glMatrixMode(GL_PROJECTION);
		glLoadIdentity();
		glOrtho(-1.0, 1.0, -1.0, 1.0, Near, Far);
		glMatrixMode(GL_MODELVIEW);
		drawPicture();
	}
	if(key == 'f')
	{
		glMatrixMode(GL_PROJECTION);
		glLoadIdentity();
		glFrustum(-0.2, 0.2, -0.2, 0.2, Near, Far);
		glMatrixMode(GL_MODELVIEW);
		drawPicture();
	}
	if(key == 'p')
	{
		if(Pause == 1) Pause = 0;
		else Pause = 1;
		drawPicture();
		terminalPrint();
	}
	if(key == 't') // Turns tracers on and off
	{
		if(Trace == 1) Trace = 0;
		else Trace = 1;
		drawPicture();
		terminalPrint();
	}
	if(key == 'M')  // Movie on/off
	{
		if(MovieFlag == 0) 
		{
			MovieFlag = 1;
			movieOn();
		}
		else 
		{
			MovieFlag = 0;
			movieOff();
		}
		terminalPrint();
	}
	
	if(key == 'S')  // Screenshot
	{	
		screenShot();
		terminalPrint();
	}

	if(key == 'C') // Center out system
	{
		zeroOutSystem();
		drawPicture();
	}
}

void mousePassiveMotionCallback(int x, int y) 
{
	// This function is called when the mouse moves without any button pressed
	// x and y are the current mouse coordinates
	
	// x and y come in as 0 to XWindowSize and 0 to YWindowSize. 
	// Use this if you choose to.
}

// This is called when you push a mouse button.
void mymouse(int button, int state, int x, int y)
{	
	if(state == GLUT_DOWN)
	{	
		if(button == GLUT_LEFT_BUTTON)
		{	
			// Do stuff in here if you choose to when the left mouse button is pressed.
		}
		else if(button == GLUT_RIGHT_BUTTON) // Right Mouse button down
		{
			// Do stuff in here if you choose to when the right mouse button is pressed.
		}
		else if(button == GLUT_MIDDLE_BUTTON)
		{
			// Do stuff in here if you choose to when the middle mouse button is pressed.
		}
	}
	
	// If no mouse button is down (state 0, they don't have a nice word like GLUT_NOT_DOWN) 
	// but you move the mouse wheel this is called.
	if(state == 0)
	{
		// When you turn the mouse whell forward this is called.
		if(button == 3)
		{
		
		}
		
		// When you turn the mouse whell backward this is called.
		else if(button == 4)
		{
		
		}
	}
}

string getTimeStamp()
{
	// Want to get a time stamp string representing current date/time, so we have a
	// unique name for each video/screenshot taken.
	time_t t = time(0); 
	struct tm * now = localtime( & t );
	int month = now->tm_mon + 1, day = now->tm_mday, year = now->tm_year, 
				curTimeHour = now->tm_hour, curTimeMin = now->tm_min, curTimeSec = now->tm_sec;
	stringstream smonth, sday, syear, stimeHour, stimeMin, stimeSec;
	smonth << month;
	sday << day;
	syear << (year + 1900); // The computer starts counting from the year 1900, so 1900 is year 0. So we fix that.
	stimeHour << curTimeHour;
	stimeMin << curTimeMin;
	stimeSec << curTimeSec;
	string timeStamp;

	if (curTimeMin <= 9)	
		timeStamp = smonth.str() + "-" + sday.str() + "-" + syear.str() + '_' + stimeHour.str() + ".0" + stimeMin.str() + 
					"." + stimeSec.str();
	else			
		timeStamp = smonth.str() + "-" + sday.str() + '-' + syear.str() + "_" + stimeHour.str() + "." + stimeMin.str() +
					"." + stimeSec.str();
	return timeStamp;
}

void movieOn()
{
	string ts = getTimeStamp();
	ts.append(".mp4");

	// Setting up the movie buffer.
	/*const char* cmd = "ffmpeg -loglevel quiet -r 60 -f rawvideo -pix_fmt rgba -s 1000x1000 -i - "
		      "-threads 0 -preset fast -y -pix_fmt yuv420p -crf 21 -vf vflip output.mp4";*/

	string baseCommand = "ffmpeg -loglevel quiet -r 60 -f rawvideo -pix_fmt rgba -s 1000x1000 -i - "
				"-c:v libx264rgb -threads 0 -preset fast -y -pix_fmt yuv420p -crf 0 -vf vflip ";

	string z = baseCommand + ts;

	const char *ccx = z.c_str();
	MovieFile = popen(ccx, "w");
	//Buffer = new int[XWindowSize*YWindowSize];
	Buffer = (int*)malloc(XWindowSize*YWindowSize*sizeof(int));
	MovieOn = 1;
}

void movieOff()
{
	if(MovieOn == 1) 
	{
		pclose(MovieFile);
	}
	free(Buffer);
	MovieOn = 0;
}

void screenShot()
{	
	int pauseFlag;
	FILE* ScreenShotFile;
	int* buffer;

	const char* cmd = "ffmpeg -loglevel quiet -framerate 60 -f rawvideo -pix_fmt rgba -s 1000x1000 -i - "
				"-c:v libx264rgb -threads 0 -preset fast -y -crf 0 -vf vflip output1.mp4";
	//const char* cmd = "ffmpeg -r 60 -f rawvideo -pix_fmt rgba -s 1000x1000 -i - "
	//              "-threads 0 -preset fast -y -pix_fmt yuv420p -crf 21 -vf vflip output1.mp4";
	ScreenShotFile = popen(cmd, "w");
	buffer = (int*)malloc(XWindowSize*YWindowSize*sizeof(int));
	
	if(Pause == 0) 
	{
		Pause = 1;
		pauseFlag = 0;
	}
	else
	{
		pauseFlag = 1;
	}
	
	for(int i =0; i < 1; i++)
	{
		drawPicture();
		glReadPixels(5, 5, XWindowSize, YWindowSize, GL_RGBA, GL_UNSIGNED_BYTE, buffer);
		fwrite(buffer, sizeof(int)*XWindowSize*YWindowSize, 1, ScreenShotFile);
	}
	
	pclose(ScreenShotFile);
	free(buffer);

	string ts = getTimeStamp(); // Only storing in a separate variable for debugging purposes.
	string s = "ffmpeg -loglevel quiet -i output1.mp4 -qscale:v 1 -qmin 1 -qmax 1 " + ts + ".jpeg";
	// Convert back to a C-style string.
	const char *ccx = s.c_str();
	system(ccx);
	system("rm output1.mp4");
	printf("\nScreenshot Captured: \n");
	cout << "Saved as " << ts << ".jpeg" << endl;

	
	//system("ffmpeg -i output1.mp4 screenShot.jpeg");
	//system("rm output1.mp4");
	
	Pause = pauseFlag;
	//ffmpeg -i output1.mp4 output_%03d.jpeg
}

void setSimulationParameters()
{
	NumberOfBodies = 16;

	TotalRunTime = 10000.0;

	Dt = 0.002;

	// This is a lennard-Jones type force G*m1*m2/(r^2) - H*m1*m2/(r^4).
	// If you want a gravity type force just set G to your gravity and set H equal 0.
	G = 0.53;

	H = 1.5;

	Epsilon = 0.01;

	MassOfBody = 1.0;

	DiameterOfBody = 0.2;

	VelocityMax = 0.0;

	Drag = 0.001;

	DrawRate = 8;
	
	PrintRate = 100;
}

void allocateMemory()
{
	BodyPositionX = (float*)malloc(NumberOfBodies*sizeof(float));
	BodyPositionY = (float*)malloc(NumberOfBodies*sizeof(float));
	BodyPositionZ = (float*)malloc(NumberOfBodies*sizeof(float));
	
	BodyVelocityX = (float*)malloc(NumberOfBodies*sizeof(float));
	BodyVelocityY = (float*)malloc(NumberOfBodies*sizeof(float));
	BodyVelocityZ = (float*)malloc(NumberOfBodies*sizeof(float));
	
	BodyForceX    = (float*)malloc(NumberOfBodies*sizeof(float));
	BodyForceY    = (float*)malloc(NumberOfBodies*sizeof(float));
	BodyForceZ    = (float*)malloc(NumberOfBodies*sizeof(float));
	
	BodyColorX    = (float*)malloc(NumberOfBodies*sizeof(float));
	BodyColorY    = (float*)malloc(NumberOfBodies*sizeof(float));
	BodyColorZ    = (float*)malloc(NumberOfBodies*sizeof(float));
}

void setInitailConditions()
{
    float dx, dy, dz, d, d2;
    int test;
	time_t t;
	float angle = 0.0;
	float dangle = 2.0*PI/NumberOfBodies;
	
	srand((unsigned) time(&t));
	for(int i = 0; i < NumberOfBodies; i++)
	{
		test = 0;
		while(test == 0)
		{
			float temp = angle + i*dangle;
			// Get random number between -1 at 1.
			// BodyPositionX[i] = ((float)rand()/(float)RAND_MAX)*2.0 - 1.0;
			// BodyPositionY[i] = ((float)rand()/(float)RAND_MAX)*2.0 - 1.0;
			// BodyPositionZ[i] = 0.0;  //((float)rand()/(float)RAND_MAX)*2.0 - 1.0;
			BodyPositionX[i] = 16*pow(sin(temp),3)/7.0;
			BodyPositionY[i] = (13*cos(temp) - 5*cos(2*temp) - 2*cos(3*temp) - cos(4*temp))/7.0;
			BodyPositionZ[i] = 0.0;
			test = 1;
			
			for(int j = 0; j < i; j++)
			{
				dx = BodyPositionX[i] - BodyPositionX[j];
				dy = BodyPositionY[i] - BodyPositionY[j];
				dz = BodyPositionZ[i] - BodyPositionZ[j];
				d2  = dx*dx + dy*dy + dz*dz;
				d = sqrt(d2);
				if(d < DiameterOfBody)
				{
					test = 0;
					break;
				}
			}
			
			if(test == 1)
			{
				BodyVelocityX[i] = 0.0; //VelocityMax*((float)rand()/(float)RAND_MAX)*2.0 - 1.0;
				BodyVelocityY[i] = 0.0; //VelocityMax*((float)rand()/(float)RAND_MAX)*2.0 - 1.0;
				BodyVelocityZ[i] = 0.0;  //VelocityMax*((float)rand()/(float)RAND_MAX)*2.0 - 1.0;
				
				BodyColorX[i] = ((float)rand()/(float)RAND_MAX);
				BodyColorY[i] = ((float)rand()/(float)RAND_MAX);
				BodyColorZ[i] = ((float)rand()/(float)RAND_MAX);
			}
		}
	}
}

float4 centerOfMass()
{
	float totalMass;
	float4 centerOfMass;
	
	centerOfMass.x = 0.0;
	centerOfMass.y = 0.0;
	centerOfMass.z = 0.0;
	totalMass = 0.0;
	
	for(int i = 0; i < NumberOfBodies; i++)
	{
    	centerOfMass.x += BodyPositionX[i]*MassOfBody;
		centerOfMass.y += BodyPositionY[i]*MassOfBody;
		centerOfMass.z += BodyPositionZ[i]*MassOfBody;
		totalMass += MassOfBody;
	}
	centerOfMass.x /= totalMass;
	centerOfMass.y /= totalMass;
	centerOfMass.z /= totalMass;
	
	return(centerOfMass);
}

float4 linearVelocity()
{
	float totalMass;
	float4 linearVelocity;
	
	linearVelocity.x = 0.0;
	linearVelocity.y = 0.0;
	linearVelocity.z = 0.0;
	totalMass = 0.0;
	
	for(int i = 0; i < NumberOfBodies; i++)
	{
    	linearVelocity.x += BodyVelocityX[i]*MassOfBody;
		linearVelocity.y += BodyVelocityY[i]*MassOfBody;
		linearVelocity.z += BodyVelocityZ[i]*MassOfBody;
		totalMass += MassOfBody;
	}
	linearVelocity.x /= totalMass;
	linearVelocity.y /= totalMass;
	linearVelocity.z /= totalMass;
	
	return(linearVelocity);
}

void zeroOutSystem()
{
	float4 pos, vel;
	pos = centerOfMass();
	vel = linearVelocity();
		
	for(int i = 0; i < NumberOfBodies; i++)
	{
		BodyPositionX[i] -= pos.x;
		BodyPositionY[i] -= pos.y;
		BodyPositionZ[i] -= pos.z;
		
		BodyVelocityX[i] -= vel.x;
		BodyVelocityY[i] -= vel.y;
		BodyVelocityZ[i] -= vel.z;
	}
}

void drawPicture()
{
	if(Trace == 0)
	{
		glClear(GL_COLOR_BUFFER_BIT);
		glClear(GL_DEPTH_BUFFER_BIT);
	}
		
	for(int i = 0; i < NumberOfBodies; i++)
	{
		BodyColorX[i] = (float)rand()/(float)RAND_MAX;
		BodyColorY[i] = (float)rand()/(float)RAND_MAX;
		BodyColorZ[i] = (float)rand()/(float)RAND_MAX;
		glColor3d(BodyColorX[i], BodyColorY[i], BodyColorZ[i]);
		glPushMatrix();
			glTranslatef(BodyPositionX[i], BodyPositionY[i], BodyPositionZ[i]);
			glutSolidSphere(DiameterOfBody/2.0, 20, 20);
		glPopMatrix();
	}
	glutSwapBuffers();
	
	if(MovieOn == 1)
	{
		glReadPixels(5, 5, XWindowSize, YWindowSize, GL_RGBA, GL_UNSIGNED_BYTE, Buffer);
		fwrite(Buffer, sizeof(int)*XWindowSize*YWindowSize, 1, MovieFile);
	}
}

void getForces(float *posX, float *posY,float *posZ, float *velX, float *velY, float *velZ, float *forceX, float *forceY, float *forceZ, float mass, float G, float H, float Epsilon, float drag, float dt, int n)
{
	float dx, dy, dz, d2, d;
	float forceMag;
    
	for(int i = 0; i < n; i++)
	{
		forceX[i] = 0.0;
		forceY[i] = 0.0;
		forceZ[i] = 0.0;
	}
	
	// Getting force
	for(int i = 0; i < n; i++)
	{   
		for(int j = i + 1; j < n; j++)
		{
			dx = posX[j] - posX[i];
			dy = posY[j] - posY[i];
			dz = posZ[j] - posZ[i];
		 	d2 = dx*dx + dy*dy + dz*dz + Epsilon;
		 	d = sqrt(d2);
			forceMag  = (G*mass*mass)/(d2) - (H*mass*mass)/(d2*d2);
			forceX[i] += forceMag*dx/d;
			forceY[i] += forceMag*dy/d;
			forceZ[i] += forceMag*dz/d;
			forceX[j] -= forceMag*dx/d;
			forceY[j] -= forceMag*dy/d;
			forceZ[j] -= forceMag*dz/d;
		}
    	}
    
    	// Updating positions
	for(int i = 0; i < n; i++)
	{
		velX[i] += ((forceX[i] - drag*velX[i])/mass)*dt;
		velY[i] += ((forceY[i] - drag*velY[i])/mass)*dt;
		velZ[i] += ((forceZ[i] - drag*velZ[i])/mass)*dt;
		
		posX[i] += velX[i]*dt;
		posY[i] += velY[i]*dt;
		posZ[i] += velZ[i]*dt;
	}
}

void nBody()
{
	if(Pause != 1)
	{	
		getForces(BodyPositionX, BodyPositionY, BodyPositionZ, BodyVelocityX, BodyVelocityY, BodyVelocityZ, BodyForceX, BodyForceY, BodyForceZ, MassOfBody,  G,  H,  Epsilon,  Drag, Dt, NumberOfBodies);
        
        	DrawTimer++;
		if(DrawTimer == DrawRate) 
		{
			drawPicture();
			DrawTimer = 0;
		}
		
		PrintTimer++;
		if(PrintTimer == PrintRate) 
		{
			terminalPrint();
			PrintTimer = 0;
		}
		
		RunTime += Dt; 
		if(TotalRunTime < RunTime)
		{
			printf("\n\n Done\n");
			exit(0);
		}
	}
}

void terminalPrint()
{
	/*
	default  \033[0m
	Black:   \033[0;30m
	Red:     \033[0;31m
	Green:   \033[0;32m
	Yellow:  \033[0;33m
	Blue:    \033[0;34m
	Magenta: \033[0;35m
	Cyan:    \033[0;36m
	White:   \033[0;37m
	printf("\033[0;30mThis text is black.\033[0m\n");
	
	BOLD_ON  "\e[1m"
	BOLD_OFF   "\e[m"
	*/
	
	system("clear");
	
	printf("\n\n");
	printf("\033[0m");
	printf(" p: Pause on/off toggle --> ");
	printf(" The simulation is:");
	if (Pause == 1) 
	{
		printf("\e[1m" " \033[0;31mPaused\n" "\e[m");
	}
	else 
	{
		printf("\e[1m" " \033[0;32mRunning\n" "\e[m");
	}
	
	printf("\n");
	printf("\033[0m");
	printf(" t: Trace on/off toggle --> ");
	printf(" Trace is:");
	if (Trace == 1) 
	{
		printf("\e[1m" " \033[0;31mOn\n" "\e[m");
	}
	else 
	{
		printf("\e[1m" " \033[0;32mOff\n" "\e[m");
	}
	
	printf("\n M: Video On/Off toggle --> ");
	if (MovieFlag == 0) 
	{
		printf("\033[0;31m");
		printf(BOLD_ON "Video Recording Off" BOLD_OFF); 
	}
	else 
	{
		printf("\033[0;32m");
		printf(BOLD_ON "Video Recording On" BOLD_OFF);
	}
	
	printf("\n");
	printf("\n S: Screenshot");

	printf("\n");
	printf("\n C: Center out system");
	
	printf("\n");
	printf("\n q: Terminates the simulation");
	
	printf("\n");
}

void setup()
{	
	setSimulationParameters();
	allocateMemory();
	setInitailConditions();
	zeroOutSystem();
    	DrawTimer = 0;
    	PrintRate = 0;
	RunTime = 0.0;
	Trace = 0;
	Pause = 1;
	MovieOn = 0;
	terminalPrint();
}

int main(int argc, char** argv)
{
	setup();
	
	XWindowSize = 1000;
	YWindowSize = 1000; 
	//Buffer = new int[XWindowSize*YWindowSize];

	// Clip plains
	Near = 0.2;
	Far = 30.0;

	//Direction here your eye is located location
	EyeX = 0.0;
	EyeY = 0.0;
	EyeZ = 2.0;

	//Where you are looking
	CenterX = 0.0;
	CenterY = 0.0;
	CenterZ = 0.0;

	//Up vector for viewing
	UpX = 0.0;
	UpY = 1.0;
	UpZ = 0.0;
	
	glutInit(&argc,argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_DEPTH | GLUT_RGB);
	glutInitWindowSize(XWindowSize,YWindowSize);
	glutInitWindowPosition(5,5);
	Window = glutCreateWindow("N Body");
	
	gluLookAt(EyeX, EyeY, EyeZ, CenterX, CenterY, CenterZ, UpX, UpY, UpZ);
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glFrustum(-0.2, 0.2, -0.2, 0.2, Near, Far);
	glMatrixMode(GL_MODELVIEW);
	glClearColor(0.0, 0.0, 0.0, 0.0);
	
	GLfloat light_position[] = {1.0, 1.0, 1.0, 0.0};
	GLfloat light_ambient[]  = {0.0, 0.0, 0.0, 1.0};
	GLfloat light_diffuse[]  = {1.0, 1.0, 1.0, 1.0};
	GLfloat light_specular[] = {1.0, 1.0, 1.0, 1.0};
	GLfloat lmodel_ambient[] = {0.2, 0.2, 0.2, 1.0};
	GLfloat mat_specular[]   = {1.0, 1.0, 1.0, 1.0};
	GLfloat mat_shininess[]  = {10.0};
	glShadeModel(GL_SMOOTH);
	glColorMaterial(GL_FRONT, GL_AMBIENT_AND_DIFFUSE);
	glLightfv(GL_LIGHT0, GL_POSITION, light_position);
	glLightfv(GL_LIGHT0, GL_AMBIENT, light_ambient);
	glLightfv(GL_LIGHT0, GL_DIFFUSE, light_diffuse);
	glLightfv(GL_LIGHT0, GL_SPECULAR, light_specular);
	glLightModelfv(GL_LIGHT_MODEL_AMBIENT, lmodel_ambient);
	glMaterialfv(GL_FRONT, GL_SPECULAR, mat_specular);
	glMaterialfv(GL_FRONT, GL_SHININESS, mat_shininess);
	glEnable(GL_LIGHTING);
	glEnable(GL_LIGHT0);
	glEnable(GL_COLOR_MATERIAL);
	glEnable(GL_DEPTH_TEST);
	
	glutPassiveMotionFunc(mousePassiveMotionCallback);
	glutMouseFunc(mymouse);
	glutDisplayFunc(Display);
	glutReshapeFunc(reshape);
	glutKeyboardFunc(KeyPressed);
	glutIdleFunc(idle);
	terminalPrint();
	glutMainLoop();
	return 0;
}






