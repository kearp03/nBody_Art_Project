#include "hip/hip_runtime.h"
// nvcc nBodyArtB.cu -o nBodyArt -lglut -lm -lGLU -lGL
//To stop hit "control c" in the window you launched it from.
#include <iostream>
#include <fstream>
#include <sstream>
#include <string.h>
#include <GL/glut.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
using namespace std;

#define PI 3.141592653589793238462643383279502884197169399375105820974944592307816406286

FILE* ffmpeg;

// defines for terminal stuff.
#define BOLD_ON  "\e[1m"
#define BOLD_OFF   "\e[m"

typedef struct {
	int id;
	float mass;
	float4 pos;
	float4 vel;
	float4 force;
	float4 color;
} Body;

FILE* MovieFile;

// Globals
int NumberOfBodies;
float TotalRunTime;
float Dt;
float G;
float H;
float dForce;
float Epsilon;
float MassOfBody;
float DiameterOfBody;
float VelocityMax;
float Drag;
int DrawRate;
int PrintRate;
// int NumberOfBodies;
int Capacity = 100;

// Other Globals
int Pause;
int LClickOn = 0;
int DrawTimer, PrintTimer;
float RunTime;
int* Buffer;
int MovieOn;
int MovieFlag;
int Trace;
double MouseX, MouseY, MouseZ;
float4 NextColor = {0.0f,0.0f,0.0f,1.0f};
string NextColorString = "Random";
int RandomColor = 1;
int GToggle = 0;
int HToggle = 0;

// Window globals
static int Window;
int XWindowSize;
int YWindowSize; 
double Near;
double Far;
double EyeX;
double EyeY;
double EyeZ;
double CenterX;
double CenterY;
double CenterZ;
double UpX;
double UpY;
double UpZ;

// Prototyping functions
void addBody(Body);
void setSimulationParameters();
void allocateMemory();
void setInitailConditions();
void drawPicture();
void nBody();
void errorCheck(const char*);
void terminalPrint();
void setup();
void movieOn();
void movieOff();
void screenShot();
float4 centerOfMass();
float4 linearVelocity();
void zeroOutSystem();

//#include "./callBackFunctions.h"

Body* Bodies = NULL;

void addBody(Body newBody) 
{
    // Reallocate memory to accommodate the new body
    if (NumberOfBodies >= Capacity) //if the new body will exceed the current capacity
    {
        Capacity *= 2; //double the capacity
        Body* temp = (Body*)realloc(Bodies, Capacity*sizeof(Body)); //reallocate memory to accommodate the new body
        if (temp == NULL)  //if memory allocation fails
        {
            fprintf(stderr, "Memory allocation failed\n");
            exit(1);
        }
        Bodies = temp;//assign the new memory to the bodies array, so long as memory allocation was successful
        //printf("Reallocated memory to capacity: %d\n", capacity);
    }

    /// Add the new body to the array
    Bodies[NumberOfBodies] = newBody;

    // Increment the number of bodies
    NumberOfBodies++;

	drawPicture();
    //for debugging
    //printf("Body %d added at (%f, %f, %f) with velocity (%f, %f, %f)\n", newBody.id, newBody.pos.x, newBody.pos.y, newBody.pos.z, newBody.vel.x, newBody.vel.y, newBody.vel.z);
}

void freeBodies() 
{
    free(Bodies);
}

void Display()
{
	glClear(GL_COLOR_BUFFER_BIT);
	glClear(GL_DEPTH_BUFFER_BIT);
	drawPicture();
}

void idle()
{
	nBody();
}

void reshape(int w, int h)
{
	glViewport(0, 0, (GLsizei) w, (GLsizei) h);
}

// void reshape(int w, int h)
// {
//     glViewport(0, 0, (GLsizei) w, (GLsizei) h);
//     glMatrixMode(GL_PROJECTION);
//     glLoadIdentity();

//     // Calculate aspect ratio
//     float aspect = (float)w / (float)h;

//     // Adjust the projection matrix based on the aspect ratio
//     if (aspect >= 1.0f) {
//         // Wider than tall
//         glOrtho(-aspect, aspect, -1.0, 1.0, Near, Far);
//     } else {
//         // Taller than wide
//         glOrtho(-1.0, 1.0, -1.0 / aspect, 1.0 / aspect, Near, Far);
//     }

//     glMatrixMode(GL_MODELVIEW);
//     glLoadIdentity();
// }

void KeyPressed(unsigned char key, int x, int y)
{	
	if(key == 'q')
	{
		// pclose(ffmpeg);
		if (ffmpeg != NULL) {
            pclose(ffmpeg);
            ffmpeg = NULL; // Optionally set to NULL after closing
        } else {
            fprintf(stderr, "Warning: Attempted to close a NULL file pointer\n");
        }
		glutDestroyWindow(Window);
		printf("\nGood Bye\n");
		exit(0);
	}
	if(key == 'o')
	{
		glMatrixMode(GL_PROJECTION);
		glLoadIdentity();
		glOrtho(-1.0, 1.0, -1.0, 1.0, Near, Far);
		glMatrixMode(GL_MODELVIEW);
		drawPicture();
	}
	if(key == 'f')
	{
		glMatrixMode(GL_PROJECTION);
		glLoadIdentity();
		glFrustum(-0.2, 0.2, -0.2, 0.2, Near, Far);
		glMatrixMode(GL_MODELVIEW);
		drawPicture();
	}
	if(key == 'p')
	{
		if(Pause == 1) Pause = 0;
		else Pause = 1;
		drawPicture();
	}
	if(key == 't') // Turns tracers on and off
	{
		if(Trace == 1) Trace = 0;
		else Trace = 1;
		drawPicture();
	}
	if(key == 'M')  // Movie on/off
	{
		if(MovieFlag == 0) 
		{
			MovieFlag = 1;
			movieOn();
		}
		else 
		{
			MovieFlag = 0;
			movieOff();
		}
	}
	
	if(key == 'S')  // Screenshot
	{	
		screenShot();
	}

	if(key == 'C') // Center out system
	{
		zeroOutSystem();
		drawPicture();
	}

	if(key == 'N') // Turn on or off adding bodies
	{
		if(LClickOn == 1) LClickOn = 0;
		else LClickOn = 1;
	}

	if(key== '0')
	{
		RandomColor = 1;
		NextColorString = "Random";
	}
	if(key== '1')
	{
		RandomColor = 0;
		NextColor={1.0f,0.4f,0.5f,1.0f};
		NextColorString = "Pink";
	}
	if(key== '2')
	{
		RandomColor = 0;
		NextColor={0.9f,1.0f,0.2f,1.0f};
		NextColorString = "Yellow";
	}
	if(key== '3')
	{
		RandomColor = 0;
		NextColor={0.9f,0.07f,0.07f,1.0f};
		NextColorString = "Red";
	}
	if(key== '4')
	{
		RandomColor = 0;
		NextColor={0.9f,0.45f,0.07f,1.0f};
		NextColorString = "Orange";
	}
	if(key== '5')
	{
		RandomColor = 0;
		NextColor={0.5f,0.92,0.4f,1.0f};
		NextColorString = "Green";
	}
	if(key== '6')
	{
		RandomColor = 0;
		NextColor={0.4f,0.69f,0.92f,1.0f};
		NextColorString = "Blue";
	}
	if(key== '7')
	{
		RandomColor = 0;
		NextColor={0.4f,0.4f,0.92f,1.0f};
		NextColorString = "Purple";
	}
	if(key== '8')
	{
		RandomColor = 0;
		NextColor={1.0f,1.0f,1.0f,1.0f};
		NextColorString = "White";
	}
	if(key== '9')
	{
		RandomColor = 0;
		NextColor={0.0f,0.0f,0.0f,1.0f};
		NextColorString = "Black";
	}
	if(key=='B'){
		NumberOfBodies--;
		drawPicture();
	}
	if(key=='R'){
		NumberOfBodies = 0;
		drawPicture();
	}
	if(key=='G'){
		if(GToggle == 0) 
		{
			GToggle = 1;
			HToggle = 0;
		}
		else GToggle = 0;
	}
	if(key=='H'){
		if(HToggle == 0)
		{
			HToggle = 1;
			GToggle = 0;
		}
		else HToggle = 0;
	}
	if(key=='['){
		if(GToggle == 1) G -= 0.1;
		if(HToggle == 1) H -= 0.01;
	}
	if(key==']'){
		if(GToggle == 1) G += 0.1;
		if(HToggle == 1) H += 0.01;
	}

	terminalPrint();
}

void mousePassiveMotionCallback(int x, int y) 
{
	// This function is called when the mouse moves without any button pressed
	// x and y are the current mouse coordinates
	
	// x and y come in as 0 to XWindowSize and 0 to YWindowSize. 
	// Use this if you choose to.
}

// This is called when you push a mouse button.
void mymouse(int button, int state, int x, int y)
{	
	if(state == GLUT_DOWN)
	{	
		if(button == GLUT_LEFT_BUTTON)
		{	
			// Do stuff in here if you choose to when the left mouse button is pressed.
			if(LClickOn == 1)
			{
				// float VelocityMax = 1.0;
				//generate random numbers for all the properties of the new body
                int index = NumberOfBodies; // Define and initialize index
				float xpos = (float)x/(float)XWindowSize*2.0 - 1.0;
				float ypos = -(float)y/(float)YWindowSize*2.0 + 1.0;
                float mass = MassOfBody;

                float colorx = ((float)rand()/(float)RAND_MAX);
                float colory = ((float)rand()/(float)RAND_MAX);
                float colorz = ((float)rand()/(float)RAND_MAX);

                Body newBody; //create a new body with the body struct

                //assign all the properties of the new body
                newBody.id = index;
				if(index == 0)
				{
					newBody.mass = 50.0;
					newBody.pos = {0.0, 0.0, 0.0, 0.0}; // Directly assign values to float4
					newBody.vel = {0.0, 0.0, 0.0, 0.0}; // Directly assign values to float4
					newBody.color = {1.0, 1.0, 1.0, 1.0f}; // Directly assign values to float4
				}
				else
				{
					newBody.mass = mass;
					newBody.pos = {xpos, ypos, 0.0f, 0.0f}; // Directly assign values to float4
					newBody.vel = {4*ypos, -4*xpos, 0.0f, 0.0f}; // Directly assign values to float4
					if(RandomColor != 1)
						newBody.color = NextColor; // Directly assign values to float4
					else
						newBody.color = {(float)rand()/(float)RAND_MAX, (float)rand()/(float)RAND_MAX, (float)rand()/(float)RAND_MAX, 1.0f}; // Directly assign values to float4

				}
                newBody.force = {0.0f, 0.0f, 0.0f, 0.0f}; // Directly assign values to float4

                addBody(newBody);
			}
		}
		else if(button == GLUT_RIGHT_BUTTON) // Right Mouse button down
		{
			// Do stuff in here if you choose to when the right mouse button is pressed.
		}
		else if(button == GLUT_MIDDLE_BUTTON)
		{
			// Do stuff in here if you choose to when the middle mouse button is pressed.
		}
	}
	
	// If no mouse button is down (state 0, they don't have a nice word like GLUT_NOT_DOWN) 
	// but you move the mouse wheel this is called.
	if(state == 0)
	{
		float dz = 0.05f;
		// When you turn the mouse wheel forward this is called.
		if(button == 3)
		{
			glTranslatef(0.0, 0.0, dz);
			drawPicture();
			terminalPrint();
		}
		
		// When you turn the mouse wheel backward this is called.
		else if(button == 4)
		{
			glTranslatef(0.0, 0.0, -dz);
			drawPicture();
			terminalPrint();
		}
	}
}

string getTimeStamp()
{
	// Want to get a time stamp string representing current date/time, so we have a
	// unique name for each video/screenshot taken.
	time_t t = time(0); 
	struct tm * now = localtime( & t );
	int month = now->tm_mon + 1, day = now->tm_mday, year = now->tm_year, 
				curTimeHour = now->tm_hour, curTimeMin = now->tm_min, curTimeSec = now->tm_sec;
	stringstream smonth, sday, syear, stimeHour, stimeMin, stimeSec;
	smonth << month;
	sday << day;
	syear << (year + 1900); // The computer starts counting from the year 1900, so 1900 is year 0. So we fix that.
	stimeHour << curTimeHour;
	stimeMin << curTimeMin;
	stimeSec << curTimeSec;
	string timeStamp;

	if (curTimeMin <= 9)	
		timeStamp = smonth.str() + "-" + sday.str() + "-" + syear.str() + '_' + stimeHour.str() + ".0" + stimeMin.str() + 
					"." + stimeSec.str();
	else			
		timeStamp = smonth.str() + "-" + sday.str() + '-' + syear.str() + "_" + stimeHour.str() + "." + stimeMin.str() +
					"." + stimeSec.str();
	return timeStamp;
}

void movieOn()
{
	string ts = getTimeStamp();
	ts.append(".mp4");

	// Setting up the movie buffer.
	/*const char* cmd = "ffmpeg -loglevel quiet -r 60 -f rawvideo -pix_fmt rgba -s 1000x1000 -i - "
		      "-threads 0 -preset fast -y -pix_fmt yuv420p -crf 21 -vf vflip output.mp4";*/

	// string baseCommand = "ffmpeg -loglevel quiet -r 60 -f rawvideo -pix_fmt rgba -s 1000x1000 -i - "
	// 			"-c:v libx264rgb -threads 0 -preset fast -y -pix_fmt yuv420p -crf 0 -vf vflip ";
	string baseCommand = "ffmpeg -loglevel quiet -r 60 -f rawvideo -pix_fmt rgba -s " + to_string(XWindowSize) + "x" + to_string(YWindowSize) + " -i - "
				"-c:v libx264rgb -threads 0 -preset fast -y -pix_fmt yuv420p -crf 0 -vf vflip ";

	string z = baseCommand + ts;

	const char *ccx = z.c_str();
	MovieFile = popen(ccx, "w");
	//Buffer = new int[XWindowSize*YWindowSize];
	Buffer = (int*)malloc(XWindowSize*YWindowSize*sizeof(int));
	MovieOn = 1;
}

void movieOff()
{
	if(MovieOn == 1) 
	{
		pclose(MovieFile);
	}
	free(Buffer);
	MovieOn = 0;
}

void screenShot()
{	
	int pauseFlag;
	FILE* ScreenShotFile;
	int* buffer;

	string tempString = "ffmpeg -loglevel quiet -framerate 60 -f rawvideo -pix_fmt rgba -s " + to_string(XWindowSize) + "x" + to_string(YWindowSize) + " -i - "
				"-c:v libx264rgb -threads 0 -preset fast -y -crf 0 -vf vflip output1.mp4";
	// const char* cmd = "ffmpeg -loglevel quiet -framerate 60 -f rawvideo -pix_fmt rgba -s 1000x1000 -i - "
	// 			"-c:v libx264rgb -threads 0 -preset fast -y -crf 0 -vf vflip output1.mp4";
	//const char* cmd = "ffmpeg -r 60 -f rawvideo -pix_fmt rgba -s 1000x1000 -i - "
	//              "-threads 0 -preset fast -y -pix_fmt yuv420p -crf 21 -vf vflip output1.mp4";
	const char* cmd = tempString.c_str();
	ScreenShotFile = popen(cmd, "w");
	buffer = (int*)malloc(XWindowSize*YWindowSize*sizeof(int));
	
	if(Pause == 0) 
	{
		Pause = 1;
		pauseFlag = 0;
	}
	else
	{
		pauseFlag = 1;
	}
	
	for(int i =0; i < 1; i++)
	{
		drawPicture();
		glReadPixels(5, 5, XWindowSize, YWindowSize, GL_RGBA, GL_UNSIGNED_BYTE, buffer);
		fwrite(buffer, sizeof(int)*XWindowSize*YWindowSize, 1, ScreenShotFile);
	}
	
	pclose(ScreenShotFile);
	free(buffer);

	string ts = getTimeStamp(); // Only storing in a separate variable for debugging purposes.
	string s = "ffmpeg -loglevel quiet -i output1.mp4 -qscale:v 1 -qmin 1 -qmax 1 " + ts + ".jpeg";
	// Convert back to a C-style string.
	const char *ccx = s.c_str();
	system(ccx);
	system("rm output1.mp4");
	printf("\nScreenshot Captured: \n");
	cout << "Saved as " << ts << ".jpeg" << endl;

	
	//system("ffmpeg -i output1.mp4 screenShot.jpeg");
	//system("rm output1.mp4");
	
	Pause = pauseFlag;
	//ffmpeg -i output1.mp4 output_%03d.jpeg
}

void setSimulationParameters()
{
	NumberOfBodies = 0;

	TotalRunTime = 10000.0;

	Dt = 0.002;

	// This is a lennard-Jones type force G*m1*m2/(r^2) - H*m1*m2/(r^4).
	// If you want a gravity type force just set G to your gravity and set H equal 0.
	G = 0.4;

	H = 0.02;

	dForce = 0.9;

	Epsilon = 0.01;

	MassOfBody = 1.0;

	DiameterOfBody = 0.1;

	VelocityMax = 0.0;

	Drag = 0.001;

	DrawRate = 8;
	
	PrintRate = 100;
}

void allocateMemory()
{
// Allocate initial memory for the bodies array
    Bodies = (Body*)malloc(Capacity*sizeof(Body));
    if (Bodies == NULL) 
    {
        fprintf(stderr, "Initial memory allocation failed\n");
        exit(1);
    }
    printf("Initial memory allocated with capacity: %d\n", Capacity);
}

void setInitailConditions()
{
    float dx, dy, dz, d, d2;
    int test;
	time_t t;
	
	srand((unsigned) time(&t));
	for(int i = 0; i < NumberOfBodies; i++)
	{
		test = 0;
		while(test == 0)
		{
			// 2D Box Shape
			// Get random number between -1 at 1.
			Bodies[i].pos.x = ((float)rand()/(float)RAND_MAX)*2.0 - 1.0;
			Bodies[i].pos.y = ((float)rand()/(float)RAND_MAX)*2.0 - 1.0;
			Bodies[i].pos.z = ((float)rand()/(float)RAND_MAX)*2.0 - 1.0; //0.0;
			// Heart Shape
			// float temp = 2*PI*((float)rand()/(float)RAND_MAX);
			// BodyPositionX[i] = 16*pow(sin(temp),3)/7.0;
			// BodyPositionY[i] = (13*cos(temp) - 5*cos(2*temp) - 2*cos(3*temp) - cos(4*temp))/7.0;
			// BodyPositionZ[i] = 0.0;
			test = 1;
			
			for(int j = 0; j < i; j++)
			{
				dx = Bodies[i].pos.x - Bodies[j].pos.x;
				dy = Bodies[i].pos.y - Bodies[j].pos.y;
				dz = Bodies[i].pos.z - Bodies[j].pos.z;
				d2  = dx*dx + dy*dy + dz*dz;
				d = sqrt(d2);
				if(d < DiameterOfBody)
				{
					test = 0;
					break;
				}
			}
			
			if(test == 1)
			{
				Bodies[i].vel.x = 0.0; //VelocityMax*((float)rand()/(float)RAND_MAX)*2.0 - 1.0;
				Bodies[i].vel.y = 0.0; //VelocityMax*((float)rand()/(float)RAND_MAX)*2.0 - 1.0;
				Bodies[i].vel.z = 0.0;  //VelocityMax*((float)rand()/(float)RAND_MAX)*2.0 - 1.0;
				
				Bodies[i].color.x = ((float)rand()/(float)RAND_MAX);
				Bodies[i].color.y = ((float)rand()/(float)RAND_MAX);
				Bodies[i].color.z = ((float)rand()/(float)RAND_MAX);
			}
		}
	}
}

float4 centerOfMass()
{
	float totalMass;
	float4 centerOfMass;
	
	centerOfMass.x = 0.0;
	centerOfMass.y = 0.0;
	centerOfMass.z = 0.0;
	totalMass = 0.0;
	
	for(int i = 0; i < NumberOfBodies; i++)
	{
    	centerOfMass.x += Bodies[i].pos.x*Bodies[i].mass;
		centerOfMass.y += Bodies[i].pos.y*Bodies[i].mass;
		centerOfMass.z += Bodies[i].pos.z*Bodies[i].mass;
		totalMass += Bodies[i].mass;
	}
	centerOfMass.x /= totalMass;
	centerOfMass.y /= totalMass;
	centerOfMass.z /= totalMass;
	
	return(centerOfMass);
}

float4 linearVelocity()
{
	float totalMass;
	float4 linearVelocity;
	
	linearVelocity.x = 0.0;
	linearVelocity.y = 0.0;
	linearVelocity.z = 0.0;
	totalMass = 0.0;
	
	for(int i = 0; i < NumberOfBodies; i++)
	{
    	linearVelocity.x += Bodies[i].vel.x*Bodies[i].mass;
		linearVelocity.y += Bodies[i].vel.y*Bodies[i].mass;
		linearVelocity.z += Bodies[i].vel.z*Bodies[i].mass;
		totalMass += Bodies[i].mass;
	}
	linearVelocity.x /= totalMass;
	linearVelocity.y /= totalMass;
	linearVelocity.z /= totalMass;
	
	return(linearVelocity);
}

void zeroOutSystem()
{
	float4 pos, vel;
	pos = centerOfMass();
	vel = linearVelocity();
		
	for(int i = 0; i < NumberOfBodies; i++)
	{
		Bodies[i].pos.x -= pos.x;
		Bodies[i].pos.y -= pos.y;
		Bodies[i].pos.z -= pos.z;
		
		Bodies[i].vel.x -= vel.x;
		Bodies[i].vel.y -= vel.y;
		Bodies[i].vel.z -= vel.z;
	}
}

void drawPicture()
{
	if(Trace == 0)
	{
		glClear(GL_COLOR_BUFFER_BIT);
		glClear(GL_DEPTH_BUFFER_BIT);
	}
		
	for(int i = 0; i < NumberOfBodies; i++)
	{
		// BodyColorX[i] = (float)rand()/(float)RAND_MAX;
		// BodyColorY[i] = (float)rand()/(float)RAND_MAX;
		// BodyColorZ[i] = (float)rand()/(float)RAND_MAX;
		glColor3d(Bodies[i].color.x, Bodies[i].color.y, Bodies[i].color.z);
		glPushMatrix();
			glTranslatef(Bodies[i].pos.x, Bodies[i].pos.y, Bodies[i].pos.z);
			glutSolidSphere(DiameterOfBody/2.0, 20, 20);
		glPopMatrix();
	}
	glutSwapBuffers();
	
	if(MovieOn == 1)
	{
		glReadPixels(5, 5, XWindowSize, YWindowSize, GL_RGBA, GL_UNSIGNED_BYTE, Buffer);
		fwrite(Buffer, sizeof(int)*XWindowSize*YWindowSize, 1, MovieFile);
	}
}

void getForces(Body* bodies, float G, float H, float Epsilon, float drag, float dt, int n)
{
	float dx, dy, dz, d2, d;
	float forceMag;
    
	for(int i = 0; i < n; i++)
	{
		bodies[i].force.x = 0.0;
		bodies[i].force.y = 0.0;
		bodies[i].force.z = 0.0;
	}
	
	// Getting force
	for(int i = 0; i < n; i++)
	{   
		for(int j = i + 1; j < n; j++)
		{
			dx = bodies[j].pos.x - bodies[i].pos.x;
			dy = bodies[j].pos.y - bodies[i].pos.y;
			dz = bodies[j].pos.z - bodies[i].pos.z;
		 	d2 = dx*dx + dy*dy + dz*dz + Epsilon;
		 	d = sqrt(d2);
			forceMag  = (G*bodies[i].mass*bodies[j].mass)/(d2) - (H*bodies[i].mass*bodies[j].mass)/(d2*d2);
			bodies[i].force.x += forceMag*dx/d;
			bodies[i].force.y += forceMag*dy/d;
			bodies[i].force.z += forceMag*dz/d;
			bodies[j].force.x -= forceMag*dx/d;
			bodies[j].force.y -= forceMag*dy/d;
			bodies[j].force.z -= forceMag*dz/d;
		}
    }
    
    // Updating positions
	for(int i = 0; i < n; i++)
	{
		bodies[i].vel.x += ((bodies[i].force.x - drag*bodies[i].vel.x)/bodies[i].mass)*dt;
		bodies[i].vel.y += ((bodies[i].force.y - drag*bodies[i].vel.y)/bodies[i].mass)*dt;
		bodies[i].vel.z += ((bodies[i].force.z - drag*bodies[i].vel.z)/bodies[i].mass)*dt;
		
		bodies[i].pos.x += bodies[i].vel.x*dt;
		bodies[i].pos.y += bodies[i].vel.y*dt;
		bodies[i].pos.z += bodies[i].vel.z*dt;
	}
	// Force changes over time
	// G *= (1 - dForce);
	// H *= (1 - dForce);

	// Move the system so that the first body is at the origin with no velocity
	for(int i = 0; i < n; i++)
	{
		bodies[i].pos.x -= bodies[0].pos.x;
		bodies[i].pos.y -= bodies[0].pos.y;
		bodies[i].pos.z -= bodies[0].pos.z;
		bodies[i].vel.x -= bodies[0].vel.x;
		bodies[i].vel.y -= bodies[0].vel.y;
		bodies[i].vel.z -= bodies[0].vel.z;
	}	
}

void nBody()
{
	if(Pause != 1)
	{	
		getForces(Bodies,  G,  H,  Epsilon,  Drag, Dt, NumberOfBodies);
        
        DrawTimer++;
		if(DrawTimer == DrawRate) 
		{
			drawPicture();
			DrawTimer = 0;
		}
		
		PrintTimer++;
		if(PrintTimer == PrintRate) 
		{
			terminalPrint();
			PrintTimer = 0;
		}
		
		RunTime += Dt; 
		if(TotalRunTime < RunTime)
		{
			printf("\n\n Done\n");
			exit(0);
		}
	}
}

void terminalPrint()
{
	/*
	default  \033[0m
	Black:   \033[0;30m
	Red:     \033[0;31m
	Green:   \033[0;32m
	Yellow:  \033[0;33m
	Blue:    \033[0;34m
	Magenta: \033[0;35m
	Cyan:    \033[0;36m
	White:   \033[0;37m
	printf("\033[0;30mThis text is black.\033[0m\n");
	
	BOLD_ON  "\e[1m"
	BOLD_OFF   "\e[m"
	*/
	
	system("clear");
	
	printf("\n\n");
	printf("\033[0m");
	printf("\n");
	printf(" o/f: Orthographic/Frustum Projection");
	printf("\n");
	printf("\n Mouse Wheel: Zoom in/out\n");
	printf("\n [] : Increase/Decrease G or H\n");
	printf("\n G: Edit G toggle --> ");
	printf(" G ");
	if (GToggle == 1) 
	{
		printf("\e[1m" " \033[0;32mCan be edited\n" "\e[m");
	}
	else 
	{
		printf("\e[1m" " \033[0;31mCannot be edited\n" "\e[m");
	}
	printf("\n G: %f", G);
	printf("\n");
	printf("\n H: Edit H toggle --> ");
	printf(" H ");
	if (HToggle == 1) 
	{
		printf("\e[1m" " \033[0;32mCan be edited\n" "\e[m");
	}
	else 
	{
		printf("\e[1m" " \033[0;31mCannot be edited\n" "\e[m");
	}
	printf("\n H: %f", H);
	printf("\n");
	printf("\n p: Pause on/off toggle --> ");
	printf(" The simulation is:");
	if (Pause == 1) 
	{
		printf("\e[1m" " \033[0;31mPaused\n" "\e[m");
	}
	else 
	{
		printf("\e[1m" " \033[0;32mRunning\n" "\e[m");
	}
	
	printf("\n");
	printf("\033[0m");
	printf(" t: Trace on/off toggle --> ");
	printf(" Trace is:");
	if (Trace == 1) 
	{
		printf("\e[1m" " \033[0;31mOn\n" "\e[m");
	}
	else 
	{
		printf("\e[1m" " \033[0;32mOff\n" "\e[m");
	}
	
	printf("\n M: Video On/Off toggle --> ");
	if (MovieFlag == 0) 
	{
		printf("\033[0;31m");
		printf(BOLD_ON "Video Recording Off" BOLD_OFF); 
	}
	else 
	{
		printf("\033[0;32m");
		printf(BOLD_ON "Video Recording On" BOLD_OFF);
	}

	printf("\n");
	printf("\n N: Add Bodies On/Off toggle --> ");
	if (LClickOn == 0) 
	{
		printf("\033[0;31m");
		printf(BOLD_ON "Adding Bodies Off" BOLD_OFF); 
	}
	else 
	{
		printf("\033[0;32m");
		printf(BOLD_ON "Adding Bodies On" BOLD_OFF);
	}
	
	printf("\n");
	printf("\n S: Screenshot");

	printf("\n");
	printf("\n C: Center out system");
	
	printf("\n");
	printf("\n q: Terminates the simulation");

	printf("\n");
	printf("\n 0: Random");
	printf("\n 1: Pink!!");
	printf("\n 2: Yellow");
	printf("\n 3: Red");
	printf("\n 4: Orange");
	printf("\n 5: Green");
	printf("\n 6: Blue");
	printf("\n 7: Purple");
	printf("\n 8: White");
	printf("\n 9: Black");
	printf("\n Next Color: %s", NextColorString.c_str());
	
	printf("\n");
	printf("\n B: Remove Last Body");
	
	printf("\n");
	printf("\n R: Remove All Bodies");
	printf("\n");
}

void setup()
{	
	setSimulationParameters();
	allocateMemory();
	setInitailConditions();
	zeroOutSystem();
    	DrawTimer = 0;
    	PrintRate = 0;
	RunTime = 0.0;
	Trace = 0;
	Pause = 1;
	MovieOn = 0;
	terminalPrint();
}

int main(int argc, char** argv)
{
	setup();
	
	XWindowSize = 1000;
	YWindowSize = 1000; 
	//Buffer = new int[XWindowSize*YWindowSize];

	// Clip plains
	Near = 0.2;
	Far = 30.0;

	//Direction here your eye is located location
	EyeX = 0.0;
	EyeY = 0.0;
	EyeZ = 2.0;

	//Where you are looking
	CenterX = 0.0;
	CenterY = 0.0;
	CenterZ = 0.0;

	//Up vector for viewing
	UpX = 0.0;
	UpY = 1.0;
	UpZ = 0.0;
	
	glutInit(&argc,argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_DEPTH | GLUT_RGB);
	glutInitWindowSize(XWindowSize,YWindowSize);
	glutInitWindowPosition(5,5);
	Window = glutCreateWindow("N Body");
	
	gluLookAt(EyeX, EyeY, EyeZ, CenterX, CenterY, CenterZ, UpX, UpY, UpZ);
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glFrustum(-0.2, 0.2, -0.2, 0.2, Near, Far);
	glMatrixMode(GL_MODELVIEW);
	glClearColor(0.0, 0.0, 0.0, 0.0);
	
	GLfloat light_position[] = {1.0, 1.0, 1.0, 0.0};
	GLfloat light_ambient[]  = {0.0, 0.0, 0.0, 1.0};
	GLfloat light_diffuse[]  = {1.0, 1.0, 1.0, 1.0};
	GLfloat light_specular[] = {1.0, 1.0, 1.0, 1.0};
	GLfloat lmodel_ambient[] = {0.2, 0.2, 0.2, 1.0};
	GLfloat mat_specular[]   = {1.0, 1.0, 1.0, 1.0};
	GLfloat mat_shininess[]  = {10.0};
	glShadeModel(GL_SMOOTH);
	glColorMaterial(GL_FRONT, GL_AMBIENT_AND_DIFFUSE);
	glLightfv(GL_LIGHT0, GL_POSITION, light_position);
	glLightfv(GL_LIGHT0, GL_AMBIENT, light_ambient);
	glLightfv(GL_LIGHT0, GL_DIFFUSE, light_diffuse);
	glLightfv(GL_LIGHT0, GL_SPECULAR, light_specular);
	glLightModelfv(GL_LIGHT_MODEL_AMBIENT, lmodel_ambient);
	glMaterialfv(GL_FRONT, GL_SPECULAR, mat_specular);
	glMaterialfv(GL_FRONT, GL_SHININESS, mat_shininess);
	glEnable(GL_LIGHTING);
	glEnable(GL_LIGHT0);
	glEnable(GL_COLOR_MATERIAL);
	glEnable(GL_DEPTH_TEST);
	
	glutPassiveMotionFunc(mousePassiveMotionCallback);
	glutMouseFunc(mymouse);
	glutDisplayFunc(Display);
	glutReshapeFunc(reshape);
	glutKeyboardFunc(KeyPressed);
	glutIdleFunc(idle);
	terminalPrint();
	glutMainLoop();
	return 0;
}